#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <omp.h>
#include <iostream>
#include <complex.h>
#include <math.h>
//#include <hipDNN.h>
#include <hip/hip_runtime.h>

const int arrSize = 4096460; // number of characters in flattenedTree.txt
//Error checking GPU calls
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define cudnnErrCheck(ans) { cudnnErrCheck_((ans), __FILE__, __LINE__); }
void cudnnErrCheck_(hipdnnStatus_t code, const char *file, int line, bool abort=true)
{
   if (code != HIPDNN_STATUS_SUCCESS) {
      fprintf(stderr, "cuDNN Error: %s %s %d\n", hipdnnGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void initGPUData_ker(float *data, int numElements, float value) {
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < numElements) {
      data[tid] = value;
   }
}

void initGPUData(float *data, int numElements, float value) {
   dim3 gridDim;
   dim3 blockDim;
   
   blockDim.x = 1024;
   gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;
   
   initGPUData_ker <<< gridDim, blockDim >>> (data, numElements, value);
}

// load information from .txt file CPU
void loadDataFromFileCPU(const char* filename, int* myDataArr)
{
    FILE *myLanguageData;
    myLanguageData = fopen(filename, "r");
    if (!myLanguageData) {
        perror("Error opening file");
        return;
    }
    
    char line[1024]; // Buffer to hold each line from the file
    int i = 0;
    while(fgets(line, sizeof(line), myLanguageData)) {
        char val1;
        int val2, val3;
        char val4;
        if(sscanf(line, "%c, %d, %d, %c", &val1, &val2, &val3, &val4) == 4) {
            // Convert characters to ASCII values and store in array
            dataArray[i++] = (int)val1;
            dataArray[i++] = val2;
            dataArray[i++] = val3;
            dataArray[i++] = (int)val4;
        }
    }

    fclose(myLanguageData);
}

using namespace std;


//function prototypes


//Part 1: Kernel Prototypes

// kernel 1:


int main(int argc, char* argv[]) {

   int seqLength;
   int numLayers;
   int hiddenSize;
   int inputSize;
   int miniBatch;
   float dropout = 0.0;
   bool bidirectional = 0;
   int mode = 2;
   int persistent = 0;

   int* myDataArr= new int [arrSize];

   loadDataFromFileCPU("../Output/flattenedTree.txt", myDataArr);


   // output data
   FILE *fp;
   fp=fopen("result.txt","w");

   // seqLength just needs to be the longest path in the training data
   // numLayers should be 8 I think
   // 
   if (argc == 5) {
      seqLength = atoi(argv[1]);
      numLayers = atoi(argv[2]);
      hiddenSize = atoi(argv[3]);
      inputSize = hiddenSize;
      miniBatch = atoi(argv[4]);
   }
   else {
      printf("Usage:\n");
      printf("./RNN <seqLength> <numLayers> <hiddenSize> <miniBatch>\n");
      return 1;
   }

   // -------------------------   
   // Create cudnn context
   // -------------------------  
   hipdnnHandle_t cudnnHandle;   
   cudnnErrCheck(hipdnnCreate(&cudnnHandle));

   
   // -------------------------   
   // Set up inputs and outputs
   // -------------------------
   
   void *x;
   void *hx = NULL;
   void *cx = NULL;
   
   void *dx;
   void *dhx = NULL;
   void *dcx = NULL;
  
   void *y;
   void *hy = NULL;
   void *cy = NULL;
   
   void *dy;
   void *dhy = NULL;
   void *dcy = NULL;
   
   // Memory allocation. hx, cx, dhx, dcx, hy, cy, dhy and dcy can be NULL.
   cudaErrCheck(hipMalloc((void**)&x, seqLength * inputSize * miniBatch * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&hx, numLayers * hiddenSize * miniBatch * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&cx, numLayers * hiddenSize * miniBatch * sizeof(float)));
   
   cudaErrCheck(hipMalloc((void**)&dx, seqLength * inputSize * miniBatch * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&dhx, numLayers * hiddenSize * miniBatch * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&dcx, numLayers * hiddenSize * miniBatch * sizeof(float)));
   
   cudaErrCheck(hipMalloc((void**)&y, seqLength * hiddenSize * miniBatch * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&hy, numLayers * hiddenSize * miniBatch * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&cy, numLayers * hiddenSize * miniBatch * sizeof(float)));
   
   cudaErrCheck(hipMalloc((void**)&dy, seqLength * hiddenSize * miniBatch * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&dhy, numLayers * hiddenSize * miniBatch * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&dcy, numLayers * hiddenSize * miniBatch * sizeof(float)));
   
   // Set up tensor descriptors. x/y/dx/dy are arrays, one per time step.
   hipdnnTensorDescriptor_t *xDesc, *yDesc, *dxDesc, *dyDesc;
   hipdnnTensorDescriptor_t hxDesc, cxDesc;
   hipdnnTensorDescriptor_t hyDesc, cyDesc;
   hipdnnTensorDescriptor_t dhxDesc, dcxDesc;
   hipdnnTensorDescriptor_t dhyDesc, dcyDesc;
   
   xDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
   yDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
   dxDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
   dyDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
   
   int dimA[3];
   int strideA[3];

   // In this example dimA[1] is constant across the whole sequence
   // This isn't required, all that is required is that it does not increase.
   for (int i = 0; i < seqLength; i++) {
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&xDesc[i]));
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&yDesc[i]));
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&dxDesc[i]));
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&dyDesc[i]));
   
      dimA[0] = miniBatch;
      dimA[1] = inputSize;
      dimA[2] = 1;
     
      strideA[0] = dimA[2] * dimA[1];
      strideA[1] = dimA[2];
      strideA[2] = 1;

      cudnnErrCheck(hipdnnSetTensorNdDescriptor(xDesc[i], HIPDNN_DATA_INT32, 3, dimA, strideA));
      cudnnErrCheck(hipdnnSetTensorNdDescriptor(dxDesc[i], HIPDNN_DATA_INT32, 3, dimA, strideA));
      
      dimA[0] = miniBatch;
      dimA[1] = hiddenSize;
      dimA[2] = 1;

      strideA[0] = dimA[2] * dimA[1];
      strideA[1] = dimA[2];
      strideA[2] = 1;
      
      cudnnErrCheck(hipdnnSetTensorNdDescriptor(yDesc[i], HIPDNN_DATA_INT32, 3, dimA, strideA));
      cudnnErrCheck(hipdnnSetTensorNdDescriptor(dyDesc[i], HIPDNN_DATA_INT32, 3, dimA, strideA));
   }
   
   
   dimA[0] = numLayers;
   dimA[1] = miniBatch;
   dimA[2] = hiddenSize;
   
   strideA[0] = dimA[2] * dimA[1];
   strideA[1] = dimA[2];
   strideA[2] = 1;
   
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&hxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&cxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&hyDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&cyDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&dhxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&dcxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&dhyDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&dcyDesc));
   
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(hxDesc, HIPDNN_DATA_INT32, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(cxDesc, HIPDNN_DATA_INT32, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(hyDesc, HIPDNN_DATA_INT32, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(cyDesc, HIPDNN_DATA_INT32, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(dhxDesc, HIPDNN_DATA_INT32, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(dcxDesc, HIPDNN_DATA_INT32, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(dhyDesc, HIPDNN_DATA_INT32, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(dcyDesc, HIPDNN_DATA_INT32, 3, dimA, strideA));
  
  
   // -------------------------
   // Set up the dropout descriptor (needed for the RNN descriptor)
   // -------------------------
   unsigned long long seed = 1337ull; // Pick a seed.
   
   hipdnnDropoutDescriptor_t dropoutDesc;
   cudnnErrCheck(hipdnnCreateDropoutDescriptor(&dropoutDesc));
   
   // How much memory does dropout need for states?
   // These states are used to generate random numbers internally
   // and should not be freed until the RNN descriptor is no longer used
   size_t stateSize;
   void *states;
   cudnnErrCheck(hipdnnDropoutGetStatesSize(cudnnHandle, &stateSize));
   
   cudaErrCheck(hipMalloc(&states, stateSize));
   
   cudnnErrCheck(hipdnnSetDropoutDescriptor(dropoutDesc, cudnnHandle, dropout, states, stateSize, seed));
                             
   // -------------------------   
   // Set up the RNN descriptor
   // -------------------------
   hipdnnRNNDescriptor_t rnnDesc;
   hipdnnRNNMode_t RNNMode;
   hipdnnRNNAlgo_t RNNAlgo;
   
   cudnnErrCheck(hipdnnCreateRNNDescriptor(&rnnDesc));
   
   RNNMode = CUDNN_RNN_LSTM;
   
   // Persistent RNNs are only supported on Pascal+ GPUs.
   if      (persistent == 0) RNNAlgo = HIPDNN_RNN_ALGO_STANDARD;
   else if (persistent == 1) RNNAlgo = HIPDNN_RNN_ALGO_PERSIST_STATIC;
   else if (persistent == 2) RNNAlgo = HIPDNN_RNN_ALGO_PERSIST_DYNAMIC;
      
   //cudnnErrCheck(hipdnnSetRNNDescriptor_v6(cudnnHandle, rnnDesc, hiddenSize, numLayers, dropoutDesc,
   //                                       HIPDNN_LINEAR_INPUT, HIPDNN_UNIDIRECTIONAL, 
   //                                       RNNMode, RNNAlgo, HIPDNN_DATA_INT32));

  cudnnErrCheck(cudnnSetRNNDescriptor_v8( rnnDesc, RNNAlgo, CUDNN_RNN_LSTM, HIPDNN_RNN_WITH_BIAS, 
                            HIPDNN_UNIDIRECTIONAL, HIPDNN_LINEAR_INPUT, HIPDNN_DATA_INT32, HIPDNN_DATA_INT32,
                            HIPDNN_DEFAULT_MATH, inputSize, hiddenSize, hiddenSize/2, numLayers, dropoutDesc, 0));
   
   
   // -------------------------
   // Set up parameters
   // -------------------------
   // This needs to be done after the rnn descriptor is set as otherwise
   // we don't know how many parameters we have to allocate
   void *w;   
   void *dw;   

   hipdnnFilterDescriptor_t wDesc, dwDesc;
   
   cudnnErrCheck(hipdnnCreateFilterDescriptor(&wDesc));
   cudnnErrCheck(hipdnnCreateFilterDescriptor(&dwDesc));
   
   size_t weightSpaceSize;
   cudnnErrCheck(cudnnGetRNNWeightSpaceSize(cudnnHandle, rnnDesc, &weightSpaceSize));
   
   int dimW[3];   
   dimW[0] =  weightSpaceSize / sizeof(int32_t);
   dimW[1] = 1;
   dimW[2] = 1;
      
   cudnnErrCheck(hipdnnSetFilterNdDescriptor(wDesc, HIPDNN_DATA_INT32, HIPDNN_TENSOR_NCHW, 3, dimW));   
   cudnnErrCheck(hipdnnSetFilterNdDescriptor(dwDesc, HIPDNN_DATA_INT32, HIPDNN_TENSOR_NCHW, 3, dimW));   
   
   cudaErrCheck(hipMalloc((void**)&w,  weightSpaceSize));
   cudaErrCheck(hipMalloc((void**)&dw, weightSpaceSize));
   
   
   // -------------------------
   // Set up work space and reserved memory
   // -------------------------   
   void *workspace;
   void *reserveSpace;   
   
   size_t workSize;
   size_t reserveSize;

   // Need for every pass
   cudnnErrCheck(hipdnnGetRNNWorkspaceSize(cudnnHandle, rnnDesc, CUDNN_FWD_MODE_TRAINING, xDesc, &workSize, &reserveSize));
   // Only needed in training, shouldn't be touched between passes.
   //cudnnErrCheck(hipdnnGetRNNWorkspaceSize(cudnnHandle, rnnDesc, CUDNN_FWD_MODE_INFERENCE, xDesc, &workSize, &reserveSize));
    
   cudaErrCheck(hipMalloc((void**)&workspace, workSize));
   cudaErrCheck(hipMalloc((void**)&reserveSpace, reserveSize));
   
   // *********************************************************************************************************
   // Initialise weights and inputs
   // *********************************************************************************************************
   // We initialise to something simple.
   // Matrices are initialised to 1 / matrixSize, biases to 1, data is 1.
   initGPUData((float*)x, seqLength * inputSize * miniBatch, 1.f);
   if (hx != NULL) initGPUData((float*)hx, numLayers * hiddenSize * miniBatch, 1.f);
   if (cx != NULL) initGPUData((float*)cx, numLayers * hiddenSize * miniBatch, 1.f);
   
   initGPUData((float*)dy, seqLength * hiddenSize * miniBatch, 1.f);
   if (dhy != NULL) initGPUData((float*)dhy, numLayers * hiddenSize * miniBatch, 1.f);
   if (dcy != NULL) initGPUData((float*)dcy, numLayers * hiddenSize * miniBatch, 1.f);
      
   
   // Weights
   // 8 Layers for LSTM.
   // 2 for traditional RNN, 6 for GRU
   int numLinearLayers = 8;
   
   for (int layer = 0; layer < numLayers; layer++) {
      for (int linLayerID = 0; linLayerID < numLinearLayers; linLayerID++) {
         
         // linear layer matrix descriptor + variable initializations
         hipdnnFilterDescriptor_t linLayerMatDesc;
         cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerMatDesc));
         int *linLayerMat;

         // linear layer bias descriptor + variable initializations
         hipdnnFilterDescriptor_t linLayerBiasDesc;
         cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerBiasDesc));
         int *linLayerBias;

         //cudnnErrCheck(hipdnnGetRNNLinLayerMatrixParams( cudnnHandle, rnnDesc, layer, xDesc[0], wDesc, w,
                                                        //linLayerID, linLayerMatDesc, (void**)&linLayerMat));
         
         hipdnnDataType_t dataType;
         hipdnnTensorFormat_t format;
         int nbDims;
         int filterDimA[3];
         // I think these calls verify that the two descriptors were made properly and can be used
         cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerMatDesc, 3, &dataType, &format, &nbDims, filterDimA));                                             
         cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerBiasDesc, 3, &dataType, &format, &nbDims, filterDimA));
                  
         

         
         // This function is used to obtain the start address and shape of every RNN weight matrix and bias vector in each pseudo-layer within the recurrent network.
         cudnnGetRNNWeightParams( cudnnHandle, rnnDesc, layer, weightSpaceSize, w, 
                                                linLayerID, linLayerMatDesc, &linLayerMat, linLayerBiasDesc, &linLayerBias);

         //cudnnErrCheck(hipdnnGetRNNLinLayerBiasParams( cudnnHandle, rnnDesc, layer, xDesc[0], wDesc, w, 
                                                      //linLayerID, linLayerBiasDesc, (void**)&linLayerBias));
         
         

         initGPUData(linLayerMat, filterDimA[0] * filterDimA[1] * filterDimA[2], 1.f / (float)(filterDimA[0] * filterDimA[1] * filterDimA[2]));                             
         initGPUData(linLayerBias, filterDimA[0] * filterDimA[1] * filterDimA[2], 1.f);

         cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerMatDesc));
         cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerBiasDesc));
      }
   }
   
   // *********************************************************************************************************
   // Dynamic persistent RNN plan (if using this algo)
   // *********************************************************************************************************
   hipdnnPersistentRNNPlan_t rnnPlan;
   if (RNNAlgo == HIPDNN_RNN_ALGO_PERSIST_DYNAMIC) {
      // Note: This step is expensive. Once completed the plan can be reused so long as the descriptor
      //       minibatch or datatype don't change.
      cudnnErrCheck(cudnnBuildRNNDynamic(cudnnHandle, rnnDesc, miniBatch));
      //cudnnErrCheck(hipdnnCreatePersistentRNNPlan(rnnDesc, miniBatch, HIPDNN_DATA_INT32, &rnnPlan));
      // Tell calls using this descriptor which plan to use.
      //cudnnErrCheck(hipdnnSetPersistentRNNPlan(rnnDesc, rnnPlan));
   }
   
   // *********************************************************************************************************
   // At this point all of the setup is done. We now need to pass through the RNN.
   // *********************************************************************************************************
   cudaErrCheck(hipDeviceSynchronize());
   
   hipEvent_t start, stop;
   float timeForward, timeBackward1, timeBackward2;
   cudaErrCheck(hipEventCreate(&start));
   cudaErrCheck(hipEventCreate(&stop));
   
   cudaErrCheck(hipEventRecord(start));   

   // If we're not training we use this instead
   /* cudnnErrCheck(hipdnnRNNForwardInference(cudnnHandle, rnnDesc, seqLength,                                          
                                             xDesc, x, hxDesc, hx, cxDesc, cx, wDesc, w, 
                                             yDesc, y, hyDesc, hy, cyDesc, cy, workspace, workSize)); */


   /*cudnnErrCheck(hipdnnRNNForwardTraining(cudnnHandle, rnnDesc, seqLength,                                       
                                         xDesc, x, hxDesc, hx, cxDesc, cx, wDesc, w, 
                                         yDesc, y, hyDesc, hy, cyDesc, cy, 
                                         workspace, workSize, reserveSpace, reserveSize));*/
   cudnnErrCheck(cudnnRNNForward( cudnnHandle, rnnDesc, CUDNN_FWD_MODE_TRAINING, NULL, xDesc, x, yDesc, y, hxDesc, hx, hy,
                    cDesc, cx, cy, weightSpaceSize, w, workSize, workSpace, reserveSize, reserveSpace));
                
   cudaErrCheck(hipEventRecord(stop));   
   cudaErrCheck(hipEventSynchronize(stop));
   cudaErrCheck(hipEventElapsedTime(&timeForward, start, stop));
   
   cudaErrCheck(hipEventRecord(start));
   

   /*cudnnErrCheck(hipdnnRNNBackwardData(cudnnHandle, rnnDesc, seqLength,                                
                               yDesc, y, dyDesc, dy, dhyDesc, dhy, dcyDesc, dcy, 
                               wDesc, w, hxDesc, hx, cxDesc, cx, dxDesc, dx, 
                               dhxDesc, dhx, dcxDesc, dcx,
                               workspace, workSize, reserveSpace, reserveSize ));*/
   cudnnErrCheck(cudnnRNNBackwardWeights_v8( cudnnHandle, rnnDesc, CUDNN_WGRAD_MODE_ADD, NULL, xDesc, x, hDesc, hx,
                               yDesc, y, weightSpaceSize, dw, workSize, workspace, reserveSize, reserveSpace));
   
   cudaErrCheck(hipEventRecord(stop));   
   cudaErrCheck(hipEventSynchronize(stop));
   cudaErrCheck(hipEventElapsedTime(&timeBackward1, start, stop));
   
   cudaErrCheck(hipEventRecord(start));
   
   // hipdnnRNNBackwardWeights adds to the data in dw.
   cudaErrCheck(hipMemset(dw, 0, weightSpaceSize));
   
   cudnnErrCheck(hipdnnRNNBackwardWeights( cudnnHandle, rnnDesc, seqLength, xDesc, x, hxDesc, hx, yDesc, y,
                                    workspace, workSize, dwDesc, dw, reserveSpace, reserveSize ));

   cudaErrCheck(hipEventRecord(stop));   

   cudaErrCheck(hipEventSynchronize(stop));
   cudaErrCheck(hipEventElapsedTime(&timeBackward2, start, stop));

   
   int numMats = 8;
   
   //if (RNNMode == HIPDNN_RNN_RELU || RNNMode == HIPDNN_RNN_TANH) {
      //numMats = 2;
   //}
   //else if (RNNMode == HIPDNN_LSTM) {
      //numMats = 8;
   //}
   //else if (RNNMode == HIPDNN_GRU) {
      //numMats = 6;
   //}
   
   // Calculate FLOPS
   printf("Forward: %3.0f GFLOPS\n", numMats * 2ull * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeForward));
   printf("Backward: %3.0f GFLOPS, ", numMats * 4ull * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * (timeBackward1 + timeBackward2)));
   printf("(%3.0f GFLOPS), ", numMats * 2ull * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeBackward1));
   printf("(%3.0f GFLOPS)\n", numMats * 2ull * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeBackward2));

   // Calculate FLOPS
   fprintf(fp,"Forward: %3.0f GFLOPS\n", numMats * 2ull * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeForward));
   fprintf(fp,"Backward: %3.0f GFLOPS, ", numMats * 4ull * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * (timeBackward1 + timeBackward2)));
   fprintf(fp,"(%3.0f GFLOPS), ", numMats * 2ull * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeBackward1));
   fprintf(fp,"(%3.0f GFLOPS)\n", numMats * 2ull * hiddenSize * hiddenSize * seqLength * miniBatch * numLayers / (1e6 * timeBackward2));

   // Make double-sure everything is finished before we copy for result checking.
   hipDeviceSynchronize();
   
   // *********************************************************************************************************
   // Print checksums.
   // *********************************************************************************************************
   /*
   if (true) {
      float* testOutputi;
      float* testOutputh;
      float* testOutputc;
      
      int biDirScale = 1;
      
      testOutputi = (float*)malloc(hiddenSize * seqLength * miniBatch * biDirScale * sizeof(float));
      testOutputh = (float*)malloc(hiddenSize * miniBatch * numLayers * biDirScale * sizeof(float));
      testOutputc = (float*)malloc(hiddenSize * miniBatch * numLayers * biDirScale * sizeof(float));
 
      cudaErrCheck(hipMemcpy(testOutputi, y, hiddenSize * seqLength * miniBatch * biDirScale * sizeof(float), hipMemcpyDeviceToHost));
      if (hy != NULL) cudaErrCheck(hipMemcpy(testOutputh, hy, numLayers * hiddenSize * miniBatch * biDirScale * sizeof(float), hipMemcpyDeviceToHost));
      if (cy != NULL && RNNMode == HIPDNN_LSTM) cudaErrCheck(hipMemcpy(testOutputc, cy, numLayers * hiddenSize * miniBatch * biDirScale * sizeof(float), hipMemcpyDeviceToHost));
      
      double checksumi = 0.f;
      double checksumh = 0.f;
      double checksumc = 0.f;
      
      for (int m = 0; m < miniBatch; m++) {
         double localSumi = 0;
         double localSumh = 0;
         double localSumc = 0;
         
         for (int j = 0; j < seqLength; j++) {
            for (int i = 0; i < hiddenSize * biDirScale; i++) {   
               localSumi += testOutputi[j * miniBatch * hiddenSize * biDirScale + m * hiddenSize * biDirScale + i];
            }
         }
         for (int j = 0; j < numLayers * biDirScale; j++) {
            for (int i = 0; i < hiddenSize; i++) {         
               if (hy != NULL) localSumh += testOutputh[j * hiddenSize * miniBatch + m * hiddenSize + i];
               if (cy != NULL) if (RNNMode == HIPDNN_LSTM) localSumc += testOutputc[j * hiddenSize * miniBatch + m * hiddenSize + i];
            }
         }
                  
         checksumi += localSumi;
         checksumh += localSumh;
         checksumc += localSumc;
      }
      
      printf("i checksum %E     ", checksumi);
      fprintf(fp,"i checksum %E     ", checksumi);
      if (RNNMode == HIPDNN_LSTM) { printf("c checksum %E     ", checksumc); fprintf(fp,"c checksum %E     ", checksumc); }
      printf("h checksum %E\n", checksumh);
      fprintf(fp,"h checksum %E\n", checksumh);
      
      free(testOutputi);
      free(testOutputc);
      free(testOutputh);
   }   
   
   if (true) {
      float* testOutputdi;
      float* testOutputdh;
      float* testOutputdc;

      int biDirScale = (bidirectional ? 2 : 1);
      
      testOutputdi = (float*)malloc(inputSize * seqLength * miniBatch * sizeof(float));
      testOutputdh = (float*)malloc(hiddenSize * miniBatch * numLayers * biDirScale * sizeof(float));
      testOutputdc = (float*)malloc(hiddenSize * miniBatch * numLayers * biDirScale * sizeof(float));
      cudaErrCheck(hipMemcpy(testOutputdi, dx, seqLength * miniBatch * inputSize * sizeof(float), hipMemcpyDeviceToHost));
      if (dhx != NULL) cudaErrCheck(hipMemcpy(testOutputdh, dhx, numLayers * hiddenSize * miniBatch * biDirScale * sizeof(float), hipMemcpyDeviceToHost));
      if (dcx != NULL) if (RNNMode == HIPDNN_LSTM) cudaErrCheck(hipMemcpy(testOutputdc, dcx, numLayers * hiddenSize * miniBatch * biDirScale * sizeof(float), hipMemcpyDeviceToHost));
      
      float checksumdi = 0.f;
      float checksumdh = 0.f;
      float checksumdc = 0.f;
      
      for (int m = 0; m < miniBatch; m++) {
         double localSumdi = 0;
         double localSumdh = 0;
         double localSumdc = 0;

         for (int j = 0; j < seqLength; j++) {
            for (int i = 0; i < inputSize; i++) {
               localSumdi += testOutputdi[j * miniBatch * inputSize + m * inputSize + i];
            }
         }

         for (int j = 0; j < numLayers * biDirScale; j++) {
            for (int i = 0; i < hiddenSize; i++) {         
               localSumdh += testOutputdh[j * hiddenSize * miniBatch + m * hiddenSize + i];
               if (RNNMode == HIPDNN_LSTM) localSumdc += testOutputdc[j * hiddenSize * miniBatch + m * hiddenSize + i];
            }
         }         

         checksumdi += localSumdi;
         checksumdh += localSumdh;
         checksumdc += localSumdc;
         
      }
      
      printf("di checksum %E    ", checksumdi);
      fprintf(fp,"di checksum %E    ", checksumdi);
      if (RNNMode == HIPDNN_LSTM) { printf("dc checksum %E    ", checksumdc); fprintf(fp,"dc checksum %E    ", checksumdc); }
      printf("dh checksum %E\n", checksumdh);
      fprintf(fp,"dh checksum %E\n", checksumdh);
      
      free(testOutputdi);
      free(testOutputdh);
      free(testOutputdc);
   }

   if (true) {
      float* testOutputdw;
      testOutputdw = (float*)malloc(weightSpaceSize);
 
      cudaErrCheck(hipMemcpy(testOutputdw, dw, weightSpaceSize, hipMemcpyDeviceToHost));
      
      double checksumdw = 0.;
            
      for (int i = 0; i < weightSpaceSize / sizeof(float); i++) {
         checksumdw += testOutputdw[i];
      }
      
      printf("dw checksum %E\n", checksumdw);
      fprintf(fp,"dw checksum %E\n", checksumdw);
      
      free(testOutputdw);
   }

  */
   if (RNNAlgo == HIPDNN_RNN_ALGO_PERSIST_DYNAMIC) {
      //hipdnnDestroyPersistentRNNPlan(rnnPlan);
   }  
  
   hipFree(x);
   hipFree(hx);
   hipFree(cx);
   hipFree(y);
   hipFree(hy);
   hipFree(cy);
   hipFree(dx);
   hipFree(dhx);
   hipFree(dcx);
   hipFree(dy);
   hipFree(dhy);
   hipFree(dcy);
   hipFree(workspace);
   hipFree(reserveSpace);
   hipFree(w);
   hipFree(dw);
   
   hipdnnDestroy(cudnnHandle);
   delete[] myDataArr;
   fclose(fp);
   return 0;
}

// Kernel Implementation