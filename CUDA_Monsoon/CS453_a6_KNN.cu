#include "hip/hip_runtime.h"
//example of running the program: ./A5_similarity_search_starter 7490 5 10 bee_dataset_1D_feature_vectors.txt

#include <hiprand.h>
#include <time.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <omp.h>
#include <iostream>
#include <complex.h>
#include <math.h>

//Error checking GPU calls
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//Define any constants here
//Feel free to change BLOCKSIZE
//#define BLOCKSIZE 128
#define DEFAULT_FILL 50.0
#define DEFAULT_MIN 1000000000.0

using namespace std;

//function prototypes
//Some of these are for debugging so I did not remove them from the starter file

void printDataset();

void warmUpGPU();

void checkParams(unsigned int N, unsigned int DIM);

void importDataset(char * fname, unsigned int N, unsigned int DIM, float * dataset);

void generateWordsCPU(float * dataset, unsigned int N, unsigned int DIM, const int NEARESTNEIGHBORS);

void calcKNN(float * distanceArray, const int NEARESTNEIGHBORS, unsigned int N, int *KNNSet, const int DIM );

bool selectRandomNeighbor(float * dataset, float * newWordArray, int * KNNSet, const int freqLocation, const int NEARESTNEIGHBORS, const int DIM);

void decodeWord(float * newWordArray, int DIM);

void outputDistanceMatrixToFile(float * distanceMatrix, unsigned int N);

void calcDistanceArrayCPU(float * dataset, float * distanceArray, float * newWordArray, const unsigned int N, const unsigned int DIM, const int wordIndex);

int translateLetter(char letterToTranslate);

int translateLetterSequel(char letterToTranslate);

//Part 1: Computing the distance matrix 

//Baseline kernel --- one thread per point/feature vector
__device__ float generateRandomNumbers(int startIdx, int endIdx);

__global__ void generateWordsBaseline(float * dataset, float * distanceArray, const unsigned int N, const unsigned int DIM);

__device__ calcKNNGPU(float * distanceArray, const int NEARESTNEIGHBORS, unsigned int N, int *KNNSet, const int DIM);
__device__ selectRandNeighborGPU(float * dataset, float * newWordArray, int * KNNSet, const int freqLocation, const int NEARESTNEIGHBORS, const int DIM);
__device__ decodeWordGPU(float * newWordArray, int DIM);
__device__ translateLetterGPU(char letterToTranslate);
__device__ calcDistanceArrayGPU(float * dataset, float * distanceArray, float * newWordArray, const unsigned int N, const unsigned int DIM, const int wordIndex);
//Part 2: querying the distance matrix
//__global__ void queryDistanceMatrixBaseline(float * distanceMatrix, const unsigned int N, const unsigned int DIM, const float epsilon, unsigned int * resultSet);


int main(int argc, char *argv[])
{
  printf("\nMODE: %d", MODE);
  warmUpGPU(); 
  //srand(time(0));
  srand(42);



  char inputFname[500];
  unsigned int N=0;
  unsigned int DIM=0;
  unsigned int NUMNEIGHBORS = 0;
  //float epsilon=0;

  if (argc != 5) {
    fprintf(stderr,"Please provide the following on the command line: N (number of lines in the file), dimensionality (number of coordinates per point), number of neighbors, dataset filename.\n");
    exit(0);
  }

  sscanf(argv[1],"%d",&N);
  sscanf(argv[2],"%d",&DIM);
  sscanf(argv[3],"%d",&NUMNEIGHBORS);
  printf("N: %d\n", N);
  printf("DIM: %d\n", DIM);
  printf("Number of neighbors: %d\n", NUMNEIGHBORS);
  strcpy(inputFname,argv[4]);

  checkParams(N, DIM);

  printf("\nAllocating the following amount of memory for the dataset: %f GiB", (sizeof(float)*N*DIM)/(1024*1024*1024.0));
  //printf("\nAllocating the following amount of memory for the distance matrix: %f GiB", (sizeof(float)*N*N)/(1024*1024*1024.0));
  

  float * dataset=(float*)malloc(sizeof(float*)*DIM*N);
  importDataset(inputFname, N, DIM, dataset);



  //CPU-only mode
  //It only computes the distance matrix but does not query the distance matrix
  if(MODE==0){
    // generate 100 words
    double tstart = omp_get_wtime();
    for (int index = 0; index < WORDSTOGENERATE; index++)
    {
      generateWordsCPU( dataset, N, DIM, NUMNEIGHBORS);
    }
    double tend = omp_get_wtime();
    printf("\nTime to compute distance matrix on the CPU: %f", tend - tstart);
    printf("\nReturning after computing on the CPU");
    return(0);
  }
  

  double tstart=omp_get_wtime();

  //Allocate memory for the dataset
  float * dev_dataset;
  gpuErrchk(hipMalloc((float**)&dev_dataset, sizeof(float)*DIM*N));
  gpuErrchk(hipMemcpy(dev_dataset, dataset, sizeof(float)*DIM*N, hipMemcpyHostToDevice));

  //For part 1 that computes the distance Array
  float * dev_distanceArray;
  gpuErrchk(hipMalloc((float**)&dev_distanceArray, sizeof(float)*N));
  

  //For part 2 for querying the distance matrix
  unsigned int * resultSet = (float *)calloc(WORDSTOGENERATE*N, sizeof(float));
  unsigned int * dev_resultSet;
  gpuErrchk(hipMalloc((float**)&dev_resultSet, sizeof(float)*N));
  gpuErrchk(hipMemcpy(dev_resultSet, resultSet, sizeof(float)*N, hipMemcpyHostToDevice));

  
  //Baseline kernels
  if(MODE==1){
  // Optimization set 1: baseline of both kernels
  unsigned int BLOCKDIM = BLOCKSIZE; 
  unsigned int NBLOCKS = NUMWORDSTOGENERATE*1.0;
  //Part 1: Compute distance matrix
  generateWordsBaseline<<<NBLOCKS, BLOCKDIM>>>(dev_dataset, dev_distanceArray, N, DIM);
  }


  //Note to reader: you can move querying the distance matrix outside of the mode
  //Part 2: Query distance matrix
  //queryDistanceMatrixBaseline<<<NBLOCKS,BLOCKDIM>>>(dev_distanceMatrix, N, DIM, epsilon, dev_resultSet);
  
  //Copy result set from the GPU
  gpuErrchk(hipMemcpy(resultSet, dev_resultSet, sizeof(unsigned int)*N, hipMemcpyDeviceToHost));

  //Compute the sum of the result set array
  unsigned int totalWithinEpsilon=0;

  //Write code here
  for (unsigned int i = 0; i < N; i++)
  {
    totalWithinEpsilon += resultSet[i];
  }
  
  printf("\nTotal number of points within epsilon: %u", totalWithinEpsilon);

  double tend=omp_get_wtime();

  printf("\n[MODE: %d, N: %d]\nTotal time: %f", MODE, N, tend-tstart);

  
  //For outputing the distance matrix for post processing (not needed for assignment --- feel free to remove)
  //float * distanceMatrix = (float*)calloc(N*N, sizeof(float));
  //gpuErrchk(hipMemcpy(distanceMatrix, dev_distanceMatrix, sizeof(float)*N*N, hipMemcpyDeviceToHost));
  //outputDistanceMatrixToFile(distanceMatrix, N);
 

  //Free memory here
  gpuErrchk(hipFree(dev_dataset));
  gpuErrchk(hipFree(dev_distanceArray));
  gpuErrchk(hipFree(dev_resultSet));
  
  free(dataset);

  printf("\n\n");
  return 0;
}


//prints the dataset that is stored in one 1-D array
void printDataset(unsigned int N, unsigned int DIM, float * dataset)
{
    for (int i=0; i<N; i++){
      for (int j=0; j<DIM; j++){
        if(j!=(DIM-1)){
          printf("%.0f,", dataset[i*DIM+j]);
        }
        else{
          printf("%.0f\n", dataset[i*DIM+j]);
        }
      }
      
    }  
}




//Import dataset as one 1-D array with N*DIM elements
//N can be made smaller for testing purposes
//DIM must be equal to the data dimensionality of the input dataset
void importDataset(char * fname, unsigned int N, unsigned int DIM, float * dataset)
{
    
    FILE *fp = fopen(fname, "r");

    if (!fp) {
        fprintf(stderr, "Unable to open file\n");
        fprintf(stderr, "Error: dataset was not imported. Returning.");
        exit(0);
    }

    unsigned int bufferSize = DIM*10;

    char buf[bufferSize];
    unsigned int rowCnt = 0;
    unsigned int colCnt = 0;
    while (fgets(buf, bufferSize, fp) && rowCnt<N) {
        // after fgets, buf should be a single line of the form a,1833,p,77,p,15,l,3,e,1
        colCnt = 0;

        char *field = strtok(buf, ",");
        char charTemp;
        int numTemp;

        sscanf(field,"%c",&charTemp);
        dataset[rowCnt*DIM+colCnt]=(float)translateLetter(charTemp);
        colCnt++;
        field = strtok(NULL, ",");
        sscanf(field,"%d",&numTemp);
        dataset[rowCnt*DIM+colCnt]=(float)numTemp;

        
        while (field) {

          colCnt++;
          field = strtok(NULL, ",");
          
          if (field!=NULL)
          {
            sscanf(field,"%c",&charTemp);
            dataset[rowCnt*DIM+colCnt]=(float)translateLetter(charTemp);
            colCnt++;
            field = strtok(NULL, ",");
            sscanf(field,"%d",&numTemp);
            dataset[rowCnt*DIM+colCnt-2] /= (float)numTemp;
            dataset[rowCnt*DIM+colCnt]=(float)numTemp;
          }   

        }
        /*
        printf("\n");
        for (int i =0; i < DIM; i++)
        {
          printf("%f ", dataset[rowCnt*DIM+i]);
        }
        printf("\n");*/

        rowCnt++;
    }

    fclose(fp);

}


void warmUpGPU(){
printf("\nWarming up GPU for time trialing...\n");
hipDeviceSynchronize();
return;
}


void checkParams(unsigned int N, unsigned int DIM)
{
  if( N <= 0 || DIM <= 0 ){
    fprintf(stderr, "\n Invalid parameters: Error, N: %u, DIM: %u", N, DIM);
    fprintf(stderr, "\nReturning");
    exit(0); 
  }
}

// Dependencies: translateLetter(), calcDistanceArrayCPU, calcKNN, selectRandomNeighbor
void generateWordsCPU(float * dataset, unsigned int N, unsigned int DIM, const int NUMNEIGHBORS)
{
  // distance array holds location values for comparison between our word and all stored words AKA N
  float * distanceArray = (float*)malloc(sizeof(float)*N);
  float * newWordArray = (float*)malloc(sizeof(float)*DIM); // random letter selected from a-z
  int newWordIdx = 1;
  float encodedLetter;
  //float nextEncodedLetter;
  double tstart = omp_get_wtime();
  int KNNSet[NUMNEIGHBORS];

 // loop through all values of KNN
  for (int i = 0; i < NUMNEIGHBORS; i++)
  {
    KNNSet[i] = 0;
  }

  // input random letter to 0 index 
  // grab a random first letter from the dataset to match
  // the frequency of letters to begin words
  // rand % N gives some value to indicate our row, *DIM to jump to the correct row, + 0 to incidate first letter
  encodedLetter =  dataset[(rand() % N)*DIM + 0];
  //char randomLetter = (rand() % 26);
  
  //translate randomLetter to int
  //encodedLetter = (float)translateLetter(randomLetter);

  // loop through new word creation
  for(int index = 0; index < DIM; index++)
  {
    //check if at beginning of word, add random letter
    if(index == 0)
    {
      newWordArray[index] = encodedLetter;
    }
    // check if at even index to add letter 
    else if(index % 2 == 0)
    {
      //newWordArray[index] = (float)(rand() % 26); 
      newWordArray[index] = (float)(rand() % 20);
    }
    // process frequencies
    else
    {
      // append '-1' to all other slots
      //newWordArray[index] = DEFAULT_FILL;
      newWordArray[index] = (float)(rand() % 10 + index);
    }
  }

  //Write code here
  bool wordEnd = false;
  for (newWordIdx = 1; newWordIdx < DIM && !wordEnd; newWordIdx++)
  {
    // LOOP THROUGH NEW WORD GEN use 
    //printf("Attempt to enter calcDistanceArrayCPU\n\n");
    calcDistanceArrayCPU(dataset, distanceArray, newWordArray, N, DIM, newWordIdx);

    // KNN function thingy takes distArr, nnearestneighbors, N
    //printf("Made it past calcDistanceArrayCPU\n\n");
    calcKNN(distanceArray, NUMNEIGHBORS, N, KNNSet, DIM);

    // print out words that correspond to the KNN idx Match
    /*for(int word = 0; word < NUMNEIGHBORS; word++)
    {
      int wordRowIdx = KNNSet[word];
      float neighbor[DIM];
      for (int index = 0; index < DIM; index++)
      {
        neighbor[index] = dataset[wordRowIdx + index];
      }
      decodeWord(neighbor, DIM);
    }*/

    // Call function to select a random neighbor from the set of K closest neighbors
    // If the random neighbor selected is _, then fill up the rest of the word with _ and end the word
    wordEnd = selectRandomNeighbor(dataset, newWordArray, KNNSet, newWordIdx, NUMNEIGHBORS, DIM);

  }


  decodeWord(newWordArray, DIM);

  free(newWordArray);
  free(distanceArray);
}

void calcKNN(float * distanceArray, const int NEARESTNEIGHBORS, unsigned int N, int *KNNSet, const int DIM )
{
  float currentMin;
  // loop through distanceArray for NEARESTNEIGHBORS iterations
  //loop from 0 to NEARESTNEIGHBORS
  for(int neighborIdx = 0; neighborIdx < NEARESTNEIGHBORS; neighborIdx++)
  {
    // set currentMin
    currentMin = DEFAULT_MIN; //1000000000.0
    // Check each distance value in distanceArr
    for(int idx = 0; idx < N; idx++)
    {
        // Check if value at distanceArr[idx] is new min
        if(distanceArray[idx] < currentMin)
        {
          //check if not on first iteration of distArr loop
          if(neighborIdx != 0)
          {
            bool neighborAlreadyAdded = false;
            // loop through already saved values in KNNSet
            for(int counter = 0; counter < neighborIdx; counter++)
            {
              //check if idx already used as a min value
              if(idx*DIM == KNNSet[counter])
              {
                // check if idx has already been used
                neighborAlreadyAdded = true;
              }
            }
            // if idx not used already
            if(!neighborAlreadyAdded)
            {
              // set val of idx in KNNSet to idx of min val in distArr
              KNNSet[neighborIdx] = idx*DIM;
            }
          }
          // on first loop of KNN
          else
          {
            // if first loop assign
            KNNSet[neighborIdx] = idx*DIM;
          }
          //update current min to new min value
          currentMin = distanceArray[idx];
        }
    }
  }
}

void calcDistanceArrayCPU(float * dataset, float * distanceArray, float * newWordArr, const unsigned int N, const unsigned int DIM, const int workingLetterIndex)
{
    //write code here
    int row = 0;
    int wordIdx = 0;
    float totalDistSquared = 0.0;
    float totalDist;

    // loop through columns
    for(row = 0; row < N; row++)
    {
      totalDistSquared = 0.0;
      //for(wordIdx = 0; wordIdx < DIM; wordIdx++)
      for(wordIdx = 0; wordIdx < workingLetterIndex + 1; wordIdx++)
      {
        totalDistSquared += (newWordArr[wordIdx]- dataset[(row * DIM )+ wordIdx]) * (newWordArr[wordIdx]-dataset[(row * DIM) + wordIdx]);
      }
      // take the sqrtf
      totalDist = sqrt(totalDistSquared);
      //then add to distanceArr
      distanceArray[row] = totalDist;
    }
}

bool selectRandomNeighbor(float * dataset, float * newWordArray, int * KNNSet, const int newLocation, const int NUMNEIGHBORS, const int DIM)
{
  bool wordEnd = false;
  // get random neighbor between 0 and 9
  int randomNeighbor = rand() % NUMNEIGHBORS;

  //int letterLocation = freqLocation + 1;
  // get index of next word from KNNSet
  /*for (int i =0; i < NUMNEIGHBORS; i++)
  {
    printf("KNN value at index %d: %d\n", i, KNNSet[i]);
  }*/

  int datasetIndex = KNNSet[randomNeighbor];

  //printf("\nAttempt to access index %d in dataset (%d + %d).\n", datasetIndex + newLocation, datasetIndex, newLocation);
  
  float newValue = dataset[datasetIndex + newLocation];
  //float newLetter = dataset[randomNeighbor*DIM + letterLocation];


  newWordArray[newLocation] = newValue;
  //newWordArray[letterLocation] = newLetter;
  if (newLocation % 2 == 0 && (int)newValue == 12) {
    wordEnd = true;
    // If word has received '_' then pad out the rest of the word with '_'
    for (int i = newLocation; i < DIM; i++)
    {
      newWordArray[i] = 12.0; // 12.0 represents '_' when decoded
    }
  }

  return wordEnd;

}

void decodeWord(float * newWordArray, int DIM)
{
  printf("\n");
  int letterIndex = -1;
  char decodeArray[27] = {'e', 't', 'a', 'o', 'i', 'n', 's',
                          'h', 'r', 'd', 'l', 'c', '_', 'u',
                          'm', 'w', 'f', 'g', 'y', 'p', 'b',
                          'v', 'k', 'j', 'x', 'q', 'z'};
  for (int index = 0; index < DIM; index += 2)
  {
    letterIndex = (int)newWordArray[index];
    if (letterIndex < 27)
    {
      printf("%c", decodeArray[letterIndex]);
    }
    else
    {
      printf("_");
    }
  }
  printf("\n");
}

//For testing/debugging
/* void computeSumOfDistances(float * distanceArray, unsigned int N)
{
  double computeSumOfDistances=0;
  for (unsigned int i=0; i<N; i++)
  {
    for (unsigned int j=0; j<N; j++)
    {
      computeSumOfDistances+=(double)distanceArray[i*N+j];
    }
  }  

  printf("\nSum of distances: %f", computeSumOfDistances);
} */

//This is used to do post-processing in Python of bee statistics
//I left it in the starter file in case anyone else wants to tinker with the 
//distance matrix and the bees, but it is unnecessary for the assignment
void outputDistanceMatrixToFile(float * distanceMatrix, unsigned int N)
{

// Open file for writing
FILE * fp = fopen( "distance_matrix_output_shared.txt", "w" ); 

 for (int i=0; i<N; i++){
    for (int j=0; j<N; j++){
      if(j!=(N-1)){
        fprintf(fp, "%.3f,", distanceMatrix[i*N+j]);
      }
      else{
        fprintf(fp, "%.3f\n", distanceMatrix[i*N+j]);
      }
    }
    
  }   

  fclose(fp);
}

int translateLetter(char letterToTranslate)
{
    int numToReturn = -1;
    switch (letterToTranslate)
    {
        case 'e':
            numToReturn = 0;
            break;
        case 't':
            numToReturn = 1;
            break;
        case 'a':
            numToReturn = 2;
            break;
        case 'o':
            numToReturn = 3;
            break;
        case 'i':
            numToReturn = 4;
            break;
        case 'n':
            numToReturn = 5;
            break;
        case 's':
            numToReturn = 6;
            break;
        case 'h':
            numToReturn = 7;
            break;
        case 'r':
            numToReturn = 8;
            break;
        case 'd':
            numToReturn = 9;
            break;
        case 'l':
            numToReturn = 10;
            break;
        case 'c':
            numToReturn = 11;
            break;
        case '_':
            numToReturn = 12;
            break;
        case 'u':
            numToReturn = 13;
            break;
        case 'm':
            numToReturn = 14;
            break;
        case 'w':
            numToReturn = 15;
            break;
        case 'f':
            numToReturn = 16;
            break;
        case 'g':
            numToReturn = 17;
            break;
        case 'y':
            numToReturn = 18;
            break;
        case 'p':
            numToReturn = 19;
            break;
        case 'b':
            numToReturn = 20;
            break;
        case 'v':
            numToReturn = 21;
            break;
        case 'k':
            numToReturn = 22;
            break;
        case 'j':
            numToReturn = 23;
            break;
        case 'x':
            numToReturn = 24;
            break;
        case 'q':
            numToReturn = 25;
            break;
        case 'z':
            numToReturn = 26;
            break;
    }
    return numToReturn;
}

int translateLetterSequel(char letterToTranslate)
{
    int numToReturn = -1;
    switch (letterToTranslate)
    {
        case 'e':
            numToReturn = 0;
            break;
        case 't':
            numToReturn = 1;
            break;
        case 'a':
            numToReturn = 2;
            break;
        case 'o':
            numToReturn = 3;
            break;
        case 'i':
            numToReturn = 4;
            break;
        case 'n':
            numToReturn = 5;
            break;
        case 's':
            numToReturn = 6;
            break;
        case 'h':
            numToReturn = 7;
            break;
        case 'r':
            numToReturn = 8;
            break;
        case 'd':
            numToReturn = 9;
            break;
        case 'l':
            numToReturn = 10;
            break;
        case 'c':
            numToReturn = 11;
            break;
        case 'u':
            numToReturn = 12;
            break;
        case 'm':
            numToReturn = 13;
            break;
        case 'w':
            numToReturn = 14;
            break;
        case 'f':
            numToReturn = 15;
            break;
        case 'g':
            numToReturn = 16;
            break;
        case 'y':
            numToReturn = 17;
            break;
        case 'p':
            numToReturn = 18;
            break;
        case 'b':
            numToReturn = 19;
            break;
        case 'v':
            numToReturn = 20;
            break;
        case 'k':
            numToReturn = 21;
            break;
        case 'j':
            numToReturn = 22;
            break;
        case 'x':
            numToReturn = 23;
            break;
        case 'q':
            numToReturn = 24;
            break;
        case 'z':
            numToReturn = 25;
            break;
        case '_':
            numToReturn = 26;
            break;
    }
    return numToReturn;
}

//Query distance matrix with one thread per feature vector
/*Once the distance matrix has been constructed, it can be queried one or
more times. In this assignment, we will perform the same query on each point in the dataset. You will
determine the number of neighbors each point has within the search distance epsilon. In other words, after
the distance matrix has been computed (Figure 3(a)) you will scan the distance matrix and compute
whether dist(p0, p1) <= epsilon, and if so, count the number of instances for each point and store it in an
array that will be returned to the host. Figure 4 shows an example of the array that will be returned
to the host. You will add the elements in this array on the host to determine the total number of
instances that points were within epsilon of each other. */

//BEGIN GPU CODE
__device__ float generateRandomNumber(hiprandState *state, int endIdx)
{
  return hiprand(state) % (endIdx + 1);
}

__global__ void generateWordsBaseline(float * dataset, float * distanceArray, const unsigned int N, const unsigned int DIM)
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if(tid < N)
  {
    // variable declaration
    float *distanceArray = new float[N]; // local var for distance array
    float * newWordArr = new float[DIM]; // array for storing new word
    int *KNNSetArr = new float[NUMNEIGHBORS]; // array to store location of neighbors

    //Compute distance array for the current point

    double tstart = omp_get_wtime();
    int newWordIdx = 1;
    float encodedLetter;

    // assign all values on KNNSetArr to 0
    if(tid < NUMNEIGHBORS)
    {
      KNNSetArr[tid] = 0;
    }

    __syncthreads();
    // generate random encoded letter from dataset
    float randomN = generateRandomNumer(state, N);
    encodedLetter = dataset[randomN * DIM + 0];

    __syncthreads();

    if(tid < DIM)
    {
      //assign random letter to zero index
      if(tid == 0)
      {
        newWordArr[0] = encodedLetter;
      }
      else if( tid % 2 == 0)
      {
        newWordArr[tid] = generateRandomNumber(state, 20); // change to var
      }
      else
      {
        newWordArr[tid] = generateRandomNumber(state, 10) + tid;
      }

      __syncthreads();

      bool wordEnd = false;
      if(!wordEnd && tid > 0)
      {
        if(newWordIdx == tid)
        {
          // need calcDistanceArrayGPU written and called

          // need calcKNNGPU written and called

          // need to set end of word and call selectRandomNeighborGPU
        }

      }
    }
  }


if(tid == 0)
{
  // implement and call decodeWordGPU pass newWordArr and DIM
}

delete[] distanceArray;
delete[] KNNSetArr;
delete[] newWordArr;
}

/*
__global__ void queryDistanceMatrixBaseline(float * distanceMatrix, const unsigned int N, const unsigned int DIM, const float epsilon, unsigned int * resultSet)
{
  //write code here
  unsigned int tid = threadIdx.x + (blockIdx.x * blockDim.x);

  if (tid < N)
  {
    resultSet[tid] = 0;
    for (unsigned int i = 0; i < N; i++)
    {
      // future modes could load in chunks of this array into shared memory
      if (distanceMatrix[tid*N+i] <= epsilon)
      {
        // future modes could use a local counter
        resultSet[tid] += 1;
      }
    }
  }
}

__global__ void queryDistanceMatrixRegisters(float * distanceMatrix, const unsigned int N, const unsigned int DIM, const float epsilon, unsigned int * resultSet)
{
  //write code here
  unsigned int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  unsigned int neighborCount = 0;

  if (tid < N)
  {
    for (unsigned int i = 0; i < N; i++)
    {
      // future modes could load in chunks of this array into shared memory
      if (distanceMatrix[i*N+tid] <= epsilon)
      {
        // future modes could use a local counter
        neighborCount++;
      }
    }
    resultSet[tid] = neighborCount;
  }
}

__global__ void queryDistanceMatrixReduction(float * distanceMatrix, const unsigned int N, const unsigned int DIM, const float epsilon, unsigned int * resultSet)
{
    //write code here
  unsigned int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  __shared__ int sharedMatrix[BLOCKSIZE];

  //Use local registers
  //unsigned int neighborCount = 0;
    // run through columns
    for (unsigned int i = 0; i < N; i++)
    {
      // check if the index in distance matrix is less than or equal to epsilon
        // tid * N gets us rows
          // + i runs us through the individual values in the row
      if (tid < N && distanceMatrix[tid*N+i] <= epsilon)
      {
        // increment count
        sharedMatrix[threadIdx.x] += 1;
      }
    }
  __syncthreads(); // finish threads writing to shared mem
  for(unsigned int stride = blockDim.x/2; stride > 0; stride /= 2)
  {
    if(threadIdx.x < stride){
      sharedMatrix[threadIdx.x] += sharedMatrix[threadIdx.x + stride];
    }
    __syncthreads();
  }
  if(threadIdx.x==0)
  {
    atomicAdd(&resultSet[tid], sharedMatrix[0]);
  }
}

//One thread per feature vector -- baseline kernel
__global__ void distanceMatrixBaseline(float * dataset, float * distanceMatrix, const unsigned int N, const unsigned int DIM)
{
  //write code here

  // each thread assigned 1 point in the dataset
  // DIM=135000
  // dataset[rowCnt*DIM+colCnt]
  
  unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
  if (tid < N)
  {
    unsigned int tidRow = tid*DIM;
    float sumOfSquaredDiffs;
    for (unsigned int otherRow = 0; otherRow < N; otherRow++)
    {
      sumOfSquaredDiffs = 0;
      for (unsigned int column = 0; column < DIM; column++)
      {
        // sum square this point's column and other point's column
        sumOfSquaredDiffs += (dataset[tidRow+column] - dataset[otherRow*DIM+column]) * (dataset[tidRow+column] - dataset[otherRow*DIM+column]);
      }
      // square root the sum up to this point + write it to distanceMatrix
      distanceMatrix[tid*N+otherRow] = sqrtf(sumOfSquaredDiffs);
    }
  }
}

// Calculate half of the distance matrix and save it for two positions of the matrix
__global__ void distanceMatrixMirrorCalculation(float * dataset, float * distanceMatrix, const unsigned int N, const unsigned int DIM)
{
  unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
  if (tid < N)
  {
    unsigned int tidRow = tid*DIM;
    float sumOfSquaredDiffs;
    for (unsigned int otherRow = tid + 1; otherRow < N; otherRow++)
    {
      sumOfSquaredDiffs = 0;
      for (unsigned int column = 0; column < DIM; column++)
      {
        // sum square this point's column and other point's column
        sumOfSquaredDiffs += (dataset[tidRow+column] - dataset[otherRow*DIM+column]) * (dataset[tidRow+column] - dataset[otherRow*DIM+column]);
      }
      // square root the sum up to this point + write it to distanceMatrix
      float totalDistance = sqrtf(sumOfSquaredDiffs);
      // mirror of this element is given by otherRow*N + tid
      distanceMatrix[tid*N+otherRow] = totalDistance;
      distanceMatrix[otherRow*N + tid] = totalDistance;
    }
  }
}

__global__ void distanceMatrixMirrorCalculationInverted(float * dataset, float * distanceMatrix, const unsigned int N, const unsigned int DIM)
{
  unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
  if (tid < N)
  {
    unsigned int tidRow = tid*DIM;
    float sumOfSquaredDiffs;
    for (unsigned int otherRow = N-1; otherRow > tid; otherRow--)
    {
      sumOfSquaredDiffs = 0;
      for (unsigned int column = 0; column < DIM; column++)
      {
        // sum square this point's column and other point's column
        sumOfSquaredDiffs += (dataset[tidRow+column] - dataset[otherRow*DIM+column]) * (dataset[tidRow+column] - dataset[otherRow*DIM+column]);
      }
      // square root the sum up to this point + write it to distanceMatrix
      float totalDistance = sqrtf(sumOfSquaredDiffs);
      // mirror of this element is given by otherRow*N + tid
      distanceMatrix[tid*N+otherRow] = totalDistance;
      distanceMatrix[otherRow*N + tid] = totalDistance;
    }
  }
}

__global__ void distanceMatrixCalculationSharedMemory(float * dataset, float * distanceMatrix, const unsigned int N, const unsigned int DIM)
{
  unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
  __shared__ int sharedMemoryBlock[WIDTH*HEIGHT];
  // declare a register array for HEIGHT other rows to accumulate to
  float sumOfSquaredDiffs[HEIGHT];
  // declare a register array for WIDTH number of ints from global dataset
  int dimensionSet[WIDTH];
  // declare shared memory array of ints for WIDTH*N elements to page into
  // enter loop to horizontally page chunks of dataset into shared memory
  for (unsigned int vStride = 0; vStride < N; vStride += HEIGHT)
  {
    // loop to reset all the sums to 0 since we're on a new set of rows
    for (unsigned int resetIndex=0; resetIndex < HEIGHT; resetIndex++)
    {
        sumOfSquaredDiffs[resetIndex] = 0.000;
    }
    // enter loop to vertically page chunks of dataset into shared memory
    for (unsigned int hStride = 0; hStride < DIM; hStride += WIDTH)
    {
      for (unsigned int registerIndex = 0; registerIndex < WIDTH; registerIndex++)
      {
        if (tid < N)
        {
          dimensionSet[registerIndex] = dataset[tid*DIM+hStride+registerIndex];
        }
      }

      // Load in a block of shared memory
      if (threadIdx.x < WIDTH)
      {
        for (unsigned int pagingIndex = 0; pagingIndex < HEIGHT; pagingIndex++)
        {
          sharedMemoryBlock[pagingIndex*WIDTH + threadIdx.x] = dataset[(vStride+pagingIndex)*DIM+hStride+threadIdx.x];
        }
      }
      __syncthreads();
      for (unsigned int otherRow = 0; otherRow < HEIGHT; otherRow++)
      {
        for (unsigned int column = 0; column < WIDTH; column++)
        {
          // sum square this point's column and other point's column
          if (tid < N)
          {
          sumOfSquaredDiffs[otherRow] += (dimensionSet[column] - sharedMemoryBlock[otherRow*WIDTH+column]) * (dimensionSet[column] - sharedMemoryBlock[otherRow*WIDTH+column]);
          }
        }
        // mirror of this element is given by otherRow*N + tid
        //distanceMatrix[tid*N+vStride+otherRow] += sumOfSquaredDiffs;
        //end of for loop for otherRow
        if (tid < N && hStride+WIDTH == DIM)
        {
          distanceMatrix[tid*N+otherRow+vStride] = sqrtf(sumOfSquaredDiffs[otherRow]);
        }

      }
      //end of for loop for horizontal stride
    } 
    //end of for loop for vertical stride
  }
}

__global__ void distanceMatrixMirrorCalculationSharedMemory(float * dataset, float * distanceMatrix, const unsigned int N, const unsigned int DIM)
{
  unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
  __shared__ int sharedMemoryBlock[WIDTH*HEIGHT];
  // declare a register array for HEIGHT other rows to accumulate to
  float sumOfSquaredDiffs[HEIGHT];
  // declare a register array for WIDTH number of ints from global dataset
  int dimensionSet[WIDTH];
  // declare shared memory array of ints for WIDTH*N elements to page into
  // enter loop to horizontally page chunks of dataset into shared memory
  for (int vStride = N-HEIGHT; vStride >= (int)(tid/HEIGHT)*HEIGHT; vStride -= HEIGHT)
  {
    // loop to reset all the sums to 0 since we're on a new set of rows
    for (unsigned int resetIndex=0; resetIndex < HEIGHT; resetIndex++)
    {
        sumOfSquaredDiffs[resetIndex] = 0.000;
    }
    // enter loop to vertically page chunks of dataset into shared memory
    for (unsigned int hStride = 0; hStride < DIM; hStride += WIDTH)
    {
      for (unsigned int registerIndex = 0; registerIndex < WIDTH; registerIndex++)
      {
        if (tid < N)
        {
          dimensionSet[registerIndex] = dataset[tid*DIM+hStride+registerIndex];
        }
      }

      // Load in a block of shared memory
      if (threadIdx.x < WIDTH)
      {
        for (unsigned int pagingIndex = 0; pagingIndex < HEIGHT; pagingIndex++)
        {
          if (tid < N)
          {
            sharedMemoryBlock[pagingIndex*WIDTH + threadIdx.x] = dataset[(vStride+pagingIndex)*DIM+hStride+threadIdx.x];
          }
        }
      }
      __syncthreads();
      for (unsigned int otherRow = 0; otherRow < HEIGHT; otherRow++)
      {
        for (unsigned int column = 0; column < WIDTH; column++)
        {
          // sum square this point's column and other point's column
          if (tid < N)
          {
          sumOfSquaredDiffs[otherRow] += (dimensionSet[column] - sharedMemoryBlock[otherRow*WIDTH+column]) * (dimensionSet[column] - sharedMemoryBlock[otherRow*WIDTH+column]);
          }
        }
        // mirror of this element is given by otherRow*N + tid
        //distanceMatrix[tid*N+vStride+otherRow] += sumOfSquaredDiffs;
        //end of for loop for otherRow
        if (tid < N && hStride+WIDTH == DIM && tid != vStride+otherRow)
        {
            distanceMatrix[(tid*N)+otherRow+vStride] = sqrtf(sumOfSquaredDiffs[otherRow]);
            distanceMatrix[(vStride+otherRow)*N+tid] = sqrtf(sumOfSquaredDiffs[otherRow]);
        }

      }
      //end of for loop for horizontal stride
    } 
    //end of for loop for vertical stride
  }
}
*/